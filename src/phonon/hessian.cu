#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Use finite difference to calculate the hessian (force constants).
    H_ij^ab = [F_i^a(-) - F_i^a(+)] / [u_j^b(+) - u_j^b(-)]
Then calculate the dynamical matrices with different k points.
------------------------------------------------------------------------------*/

#include "force/force.cuh"
#include "force/force_constant.cuh"
#include "hessian.cuh"
#include "utilities/common.cuh"
#include "utilities/cusolver_wrapper.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/read_file.cuh"
#include <vector>
#include <cstring>
#include <fstress>
#include <map>

namespace {
const std::map<std::string, double> table = {
  {"H", 1.0080000000}, {"He", 4.0026020000}, {"Li", 6.9400000000}, {"Be", 9.0121831000},
  {"B", 10.8100000000}, {"C", 12.0110000000}, {"N", 14.0070000000}, {"O", 15.9990000000},
  {"F", 18.9984031630}, {"Ne", 20.1797000000}, {"Na", 22.9897692800}, {"Mg", 24.3050000000},
  {"Al", 26.9815385000}, {"Si", 28.0850000000}, {"P", 30.9737619980}, {"S", 32.0600000000},
  {"Cl", 35.4500000000}, {"Ar", 39.9480000000}, {"K", 39.0983000000}, {"Ca", 40.0780000000},
  {"Sc", 44.9559080000}, {"Ti", 47.8670000000}, {"V", 50.9415000000}, {"Cr", 51.9961000000},
  {"Mn", 54.9380440000}, {"Fe", 55.8450000000}, {"Co", 58.9331940000}, {"Ni", 58.6934000000},
  {"Cu", 63.5460000000}, {"Zn", 65.3800000000}, {"Ga", 69.7230000000}, {"Ge", 72.6300000000},
  {"As", 74.9215950000}, {"Se", 78.9710000000}, {"Br", 79.9040000000}, {"Kr", 83.7980000000},
  {"Rb", 85.4678000000}, {"Sr", 87.6200000000}, {"Y", 88.9058400000}, {"Zr", 91.2240000000},
  {"Nb", 92.9063700000}, {"Mo", 95.9500000000}, {"Tc", 98}, {"Ru", 101.0700000000},
  {"Rh", 102.9055000000}, {"Pd", 106.4200000000}, {"Ag", 107.8682000000}, {"Cd", 112.4140000000},
  {"In", 114.8180000000}, {"Sn", 118.7100000000}, {"Sb", 121.7600000000}, {"Te", 127.6000000000},
  {"I", 126.9044700000}, {"Xe", 131.2930000000}, {"Cs", 132.9054519600}, {"Ba", 137.3270000000},
  {"La", 138.9054700000}, {"Ce", 140.1160000000}, {"Pr", 140.9076600000}, {"Nd", 144.2420000000},
  {"Pm", 145}, {"Sm", 150.3600000000}, {"Eu", 151.9640000000}, {"Gd", 157.2500000000},
  {"Tb", 158.9253500000}, {"Dy", 162.5000000000}, {"Ho", 164.9303300000}, {"Er", 167.2590000000},
  {"Tm", 168.9342200000}, {"Yb", 173.0450000000}, {"Lu", 174.9668000000}, {"Hf", 178.4900000000},
  {"Ta", 180.9478800000}, {"W", 183.8400000000}, {"Re", 186.2070000000}, {"Os", 190.2300000000},
  {"Ir", 192.2170000000}, {"Pt", 195.0840000000}, {"Au", 196.9665690000}, {"Hg", 200.5920000000},
  {"Tl", 204.3800000000}, {"Pb", 207.2000000000}, {"Bi", 208.9804000000}, {"Po", 210}, {"At", 210},
  {"Rn", 222}, {"Fr", 223}, {"Ra", 226}, {"Ac", 227}, {"Th", 232.0377000000}, {"Pa", 231.0358800000},
  {"U", 238.0289100000}, {"Np", 237}, {"Pu", 244}, {"Am", 243}, {"Cm", 247}, {"Bk", 247}, {"Cf", 251},
  {"Es", 252}, {"Fm", 257}, {"Md", 258}, {"No", 259}, {"Lr", 262}
};
double mass_of(const std::string& sym) {
  auto it = table.find(sym);
  if (it == table.end()) {
      PRINT_INPUT_ERROR("Error: no such element '" << sym << "'\n");
  }
  return it->second;
} 
}

void Hessian::compute(
  Atom& atoms,
  Force& force,
  Box& box,
  std::vector<double>& cpu_position_per_atom,
  GPU_Vector<double>& position_per_atom,
  GPU_Vector<int>& type,
  std::vector<Group>& group,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  initialize(Atom& atoms, type.size());
  find_H(
    force,
    box,
    cpu_position_per_atom,
    position_per_atom,
    type,
    group,
    potential_per_atom,
    force_per_atom,
    virial_per_atom);

  if (num_kpoints == 1) // currently for Alex's GKMA calculations
  {
    find_D(box, cpu_position_per_atom);
    find_eigenvectors();
  } else {
    find_dispersion(box, cpu_position_per_atom);
  }
}

void Hessian::create_basis(Atom& atoms, size_t N)
{
  std::ifstream fin("run.in");
  std::string key;
  size_t cx, cy, cz;
  if (!(fin >> key >> cx >> cy >> cz) || key != "replicate")
    PRINT_INPUT_ERROR("replicate is required in run.in\n");
  const size_t cx = r[0], cy = r[1], cz = r[2];
  size_t num_basis = N / (cx * cy * cz);

  basis.resize(num_basis);
  mass.resize(num_basis);
  for (size_t i = 0; i < num_basis; ++i) {
    basis[i] = atoms.cpu_atom_symbol[i];
    mass[i]  = mass_of(atoms.cpu_atom_symbol[i]);
  }

  label.resize(N);
  size_t idx = 0;
  for (size_t cell = 0; cell < cx * cy * cz; ++cell)
    for (size_t atom = 0; atom < num_basis; ++atom)
      label[idx++] = atom;
}

void Hessian::read_kpoints()
{
  FILE* fid = fopen("kpoints.in", "r");
  size_t count;
  count = fscanf(fid, "%zu", &num_kpoints);
  PRINT_SCANF_ERROR(count, 1, "Reading error for kpoints.in.");

  kpoints.resize(num_kpoints * 3);
  for (size_t m = 0; m < num_kpoints; ++m) {
    count = fscanf(fid, "%lf%lf%lf", &kpoints[m * 3 + 0], &kpoints[m * 3 + 1], &kpoints[m * 3 + 2]);
    PRINT_SCANF_ERROR(count, 3, "Reading error for kpoints.in.");
  }
  fclose(fid);
}

void Hessian::initialize(Atom& atoms, size_t N)
{
  create_basis(atoms, N);
  read_kpoints();
  size_t num_H = num_basis * N * 9;
  size_t num_D = num_basis * num_basis * 9 * num_kpoints;
  H.resize(num_H, 0.0);
  DR.resize(num_D, 0.0);
  if (num_kpoints > 1) // for dispersion calculation
  {
    DI.resize(num_D, 0.0);
  }
}

bool Hessian::is_too_far(
  const Box& box,
  const std::vector<double>& cpu_position_per_atom,
  const size_t n1,
  const size_t n2)
{
  const int number_of_atoms = cpu_position_per_atom.size() / 3;
  double x12 = cpu_position_per_atom[n2] - cpu_position_per_atom[n1];
  double y12 =
    cpu_position_per_atom[n2 + number_of_atoms] - cpu_position_per_atom[n1 + number_of_atoms];
  double z12 = cpu_position_per_atom[n2 + number_of_atoms * 2] -
               cpu_position_per_atom[n1 + number_of_atoms * 2];
  apply_mic(box, x12, y12, z12);
  double d12_square = x12 * x12 + y12 * y12 + z12 * z12;
  return (d12_square > (cutoff * cutoff));
}

void Hessian::find_H(
  Force& force,
  Box& box,
  std::vector<double>& cpu_position_per_atom,
  GPU_Vector<double>& position_per_atom,
  GPU_Vector<int>& type,
  std::vector<Group>& group,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();

  for (size_t nb = 0; nb < num_basis; ++nb) {
    size_t n1 = basis[nb];
    for (size_t n2 = 0; n2 < number_of_atoms; ++n2) {
      if (is_too_far(box, cpu_position_per_atom, n1, n2)) {
        continue;
      }
      size_t offset = (nb * number_of_atoms + n2) * 9;
      find_H12(
        displacement,
        n1,
        n2,
        box,
        position_per_atom,
        type,
        group,
        potential_per_atom,
        force_per_atom,
        virial_per_atom,
        force,
        H.data() + offset);
    }
  }
}

static void find_exp_ikr(
  const size_t n1,
  const size_t n2,
  const double* k,
  const Box& box,
  const std::vector<double>& cpu_position_per_atom,
  double& cos_kr,
  double& sin_kr)
{
  const int number_of_atoms = cpu_position_per_atom.size() / 3;
  double x12 = cpu_position_per_atom[n2] - cpu_position_per_atom[n1];
  double y12 =
    cpu_position_per_atom[n2 + number_of_atoms] - cpu_position_per_atom[n1 + number_of_atoms];
  double z12 = cpu_position_per_atom[n2 + number_of_atoms * 2] -
               cpu_position_per_atom[n1 + number_of_atoms * 2];
  apply_mic(box, x12, y12, z12);
  double kr = k[0] * x12 + k[1] * y12 + k[2] * z12;
  cos_kr = cos(kr);
  sin_kr = sin(kr);
}

void Hessian::output_D()
{
  FILE* fid = fopen("D.out", "w");
  for (size_t nk = 0; nk < num_kpoints; ++nk) {
    size_t offset = nk * num_basis * num_basis * 9;
    for (size_t n1 = 0; n1 < num_basis * 3; ++n1) {
      for (size_t n2 = 0; n2 < num_basis * 3; ++n2) {
        // cuSOLVER requires column-major
        fprintf(fid, "%g ", DR[offset + n1 + n2 * num_basis * 3]);
      }
      if (num_kpoints > 1) {
        for (size_t n2 = 0; n2 < num_basis * 3; ++n2) {
          // cuSOLVER requires column-major
          fprintf(fid, "%g ", DI[offset + n1 + n2 * num_basis * 3]);
        }
      }
      fprintf(fid, "\n");
    }
  }
  fclose(fid);
}

void Hessian::find_omega(FILE* fid, size_t offset)
{
  size_t dim = num_basis * 3;
  std::vector<double> W(dim);
  eig_hermitian_QR(dim, DR.data() + offset, DI.data() + offset, W.data());
  double natural_to_THz = 1.0e6 / (TIME_UNIT_CONVERSION * TIME_UNIT_CONVERSION);
  for (size_t n = 0; n < dim; ++n) {
    fprintf(fid, "%g ", W[n] * natural_to_THz);
  }
  fprintf(fid, "\n");
}

void Hessian::find_omega_batch(FILE* fid)
{
  size_t dim = num_basis * 3;
  std::vector<double> W(dim * num_kpoints);
  eig_hermitian_Jacobi_batch(dim, num_kpoints, DR.data(), DI.data(), W.data());
  double natural_to_THz = 1.0e6 / (TIME_UNIT_CONVERSION * TIME_UNIT_CONVERSION);
  for (size_t nk = 0; nk < num_kpoints; ++nk) {
    size_t offset = nk * dim;
    for (size_t n = 0; n < dim; ++n) {
      fprintf(fid, "%g ", W[offset + n] * natural_to_THz);
    }
    fprintf(fid, "\n");
  }
}

void Hessian::find_dispersion(const Box& box, const std::vector<double>& cpu_position_per_atom)
{
  const int number_of_atoms = cpu_position_per_atom.size() / 3;

  FILE* fid_omega2 = fopen("omega2.out", "w");
  for (size_t nk = 0; nk < num_kpoints; ++nk) {
    size_t offset = nk * num_basis * num_basis * 9;
    for (size_t nb = 0; nb < num_basis; ++nb) {
      size_t n1 = basis[nb];
      size_t label_1 = label[n1];
      double mass_1 = mass[label_1];
      for (size_t n2 = 0; n2 < number_of_atoms; ++n2) {
        if (is_too_far(box, cpu_position_per_atom, n1, n2))
          continue;
        double cos_kr, sin_kr;
        find_exp_ikr(n1, n2, kpoints.data() + nk * 3, box, cpu_position_per_atom, cos_kr, sin_kr);

        size_t label_2 = label[n2];
        double mass_2 = mass[label_2];
        double mass_factor = 1.0 / sqrt(mass_1 * mass_2);
        double* H12 = H.data() + (nb * number_of_atoms + n2) * 9;
        for (size_t a = 0; a < 3; ++a) {
          for (size_t b = 0; b < 3; ++b) {
            size_t a3b = a * 3 + b;
            size_t row = label_1 * 3 + a;
            size_t col = label_2 * 3 + b;
            // cuSOLVER requires column-major
            size_t index = offset + col * num_basis * 3 + row;
            DR[index] += H12[a3b] * cos_kr * mass_factor;
            DI[index] += H12[a3b] * sin_kr * mass_factor;
          }
        }
      }
    }
    if (num_basis > 10) {
      find_omega(fid_omega2, offset);
    } // > 32x32
  }
  output_D();
  if (num_basis <= 10) {
    find_omega_batch(fid_omega2);
  } // <= 32x32
  fclose(fid_omega2);
}

void Hessian::find_D(const Box& box, std::vector<double>& cpu_position_per_atom)
{
  const int number_of_atoms = cpu_position_per_atom.size() / 3;

  for (size_t nb = 0; nb < num_basis; ++nb) {
    size_t n1 = basis[nb];
    size_t label_1 = label[n1];
    double mass_1 = mass[label_1];
    for (size_t n2 = 0; n2 < number_of_atoms; ++n2) {
      if (is_too_far(box, cpu_position_per_atom, n1, n2)) {
        continue;
      }

      size_t label_2 = label[n2];
      double mass_2 = mass[label_2];
      double mass_factor = 1.0 / sqrt(mass_1 * mass_2);
      double* H12 = H.data() + (nb * number_of_atoms + n2) * 9;
      for (size_t a = 0; a < 3; ++a) {
        for (size_t b = 0; b < 3; ++b) {
          size_t a3b = a * 3 + b;
          size_t row = label_1 * 3 + a;
          size_t col = label_2 * 3 + b;
          // cuSOLVER requires column-major
          size_t index = col * num_basis * 3 + row;
          DR[index] += H12[a3b] * mass_factor;
        }
      }
    }
  }
}

void Hessian::find_eigenvectors()
{
  std::ofstream eigfile;
  eigfile.open("eigenvector.out", std::ios::out | std::ios::binary);

  size_t dim = num_basis * 3;
  std::vector<double> W(dim);
  std::vector<double> eigenvectors(dim * dim);
  eigenvectors_symmetric_Jacobi(dim, DR.data(), W.data(), eigenvectors.data());

  double natural_to_THz = 1.0e6 / (TIME_UNIT_CONVERSION * TIME_UNIT_CONVERSION);

  // output eigenvalues
  float om2;
  for (size_t n = 0; n < dim; n++) {
    om2 = (float)(W[n] * natural_to_THz);
    eigfile.write((char*)&om2, sizeof(float));
  }

  // output eigenvectors
  float eig;
  for (size_t col = 0; col < dim; col++) {
    for (size_t a = 0; a < 3; a++) {
      for (size_t b = 0; b < num_basis; b++) {
        size_t row = a + b * 3;
        // column-major order from cuSolver
        eig = (float)eigenvectors[row + col * dim];
        eigfile.write((char*)&eig, sizeof(float));
      }
    }
  }
  eigfile.close();
}

void Hessian::parse(const char** param, size_t num_param)
{
  if (num_param != 3) {
    PRINT_INPUT_ERROR("compute_phonon should have 2 parameters.\n");
  }
  // cutoff
  if (!is_valid_real(param[1], &cutoff)) {
    PRINT_INPUT_ERROR("cutoff for compute_phonon should be a number.\n");
  }
  if (cutoff <= 0) {
    PRINT_INPUT_ERROR("cutoff for compute_phonon should be positive.\n");
  }
  printf("Cutoff distance for compute_phonon = %g A.\n", cutoff);

  // displacement
  if (!is_valid_real(param[2], &displacement)) {
    PRINT_INPUT_ERROR("displacement for compute_phonon should be a number.\n");
  }
  if (displacement <= 0) {
    PRINT_INPUT_ERROR("displacement for compute_phonon should be positive.\n");
  }
  printf("displacement for compute_phonon = %g A.\n", displacement);
}

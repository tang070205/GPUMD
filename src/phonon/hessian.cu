#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Use finite difference to calculate the hessian (force constants).
    H_ij^ab = [F_i^a(-) - F_i^a(+)] / [u_j^b(+) - u_j^b(-)]
Then calculate the dynamical matrices with different k points.
------------------------------------------------------------------------------*/

#include "force/force.cuh"
#include "force/force_constant.cuh"
#include "hessian.cuh"
#include "utilities/common.cuh"
#include "utilities/cusolver_wrapper.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/read_file.cuh"
#include <vector>
#include <cstring>
#include <map>
#include <cmath>
#include <sstream>
#include <string>

const std::map<std::string, double> table = {
  {"H", 1.0080000000}, {"He", 4.0026020000}, {"Li", 6.9400000000}, {"Be", 9.0121831000},
  {"B", 10.8100000000}, {"C", 12.0110000000}, {"N", 14.0070000000}, {"O", 15.9990000000},
  {"F", 18.9984031630}, {"Ne", 20.1797000000}, {"Na", 22.9897692800}, {"Mg", 24.3050000000},
  {"Al", 26.9815385000}, {"Si", 28.0850000000}, {"P", 30.9737619980}, {"S", 32.0600000000},
  {"Cl", 35.4500000000}, {"Ar", 39.9480000000}, {"K", 39.0983000000}, {"Ca", 40.0780000000},
  {"Sc", 44.9559080000}, {"Ti", 47.8670000000}, {"V", 50.9415000000}, {"Cr", 51.9961000000},
  {"Mn", 54.9380440000}, {"Fe", 55.8450000000}, {"Co", 58.9331940000}, {"Ni", 58.6934000000},
  {"Cu", 63.5460000000}, {"Zn", 65.3800000000}, {"Ga", 69.7230000000}, {"Ge", 72.6300000000},
  {"As", 74.9215950000}, {"Se", 78.9710000000}, {"Br", 79.9040000000}, {"Kr", 83.7980000000},
  {"Rb", 85.4678000000}, {"Sr", 87.6200000000}, {"Y", 88.9058400000}, {"Zr", 91.2240000000},
  {"Nb", 92.9063700000}, {"Mo", 95.9500000000}, {"Tc", 98}, {"Ru", 101.0700000000},
  {"Rh", 102.9055000000}, {"Pd", 106.4200000000}, {"Ag", 107.8682000000}, {"Cd", 112.4140000000},
  {"In", 114.8180000000}, {"Sn", 118.7100000000}, {"Sb", 121.7600000000}, {"Te", 127.6000000000},
  {"I", 126.9044700000}, {"Xe", 131.2930000000}, {"Cs", 132.9054519600}, {"Ba", 137.3270000000},
  {"La", 138.9054700000}, {"Ce", 140.1160000000}, {"Pr", 140.9076600000}, {"Nd", 144.2420000000},
  {"Pm", 145}, {"Sm", 150.3600000000}, {"Eu", 151.9640000000}, {"Gd", 157.2500000000},
  {"Tb", 158.9253500000}, {"Dy", 162.5000000000}, {"Ho", 164.9303300000}, {"Er", 167.2590000000},
  {"Tm", 168.9342200000}, {"Yb", 173.0450000000}, {"Lu", 174.9668000000}, {"Hf", 178.4900000000},
  {"Ta", 180.9478800000}, {"W", 183.8400000000}, {"Re", 186.2070000000}, {"Os", 190.2300000000},
  {"Ir", 192.2170000000}, {"Pt", 195.0840000000}, {"Au", 196.9665690000}, {"Hg", 200.5920000000},
  {"Tl", 204.3800000000}, {"Pb", 207.2000000000}, {"Bi", 208.9804000000}, {"Po", 210}, {"At", 210},
  {"Rn", 222}, {"Fr", 223}, {"Ra", 226}, {"Ac", 227}, {"Th", 232.0377000000}, {"Pa", 231.0358800000},
  {"U", 238.0289100000}, {"Np", 237}, {"Pu", 244}, {"Am", 243}, {"Cm", 247}, {"Bk", 247}, {"Cf", 251},
  {"Es", 252}, {"Fm", 257}, {"Md", 258}, {"No", 259}, {"Lr", 262}
};

void Hessian::compute(
  Force& force,
  Box& box,
  std::vector<std::string> cpu_atom_symbol,
  std::vector<double>& cpu_position_per_atom,
  GPU_Vector<double>& position_per_atom,
  GPU_Vector<int>& type,
  std::vector<Group>& group,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  initialize(cpu_atom_symbol, box, type.size());
  find_H(
    force,
    box,
    cpu_position_per_atom,
    position_per_atom,
    type,
    group,
    potential_per_atom,
    force_per_atom,
    virial_per_atom);

  if (num_kpoints == 1) // currently for Alex's GKMA calculations
  {
    find_D(box, cpu_position_per_atom);
    find_eigenvectors();
  } else {
    find_dispersion(box, cpu_position_per_atom);
  }
}

void Hessian::create_basis(std::vector<std::string> cpu_atom_symbol, size_t N)
{
  std::ifstream fin("run.in");
  std::string key;
  if (fin >> key && key == "replicate")
    fin >> cx >> cy >> cz;
  this->num_basis = N / (cx * cy * cz);

  basis.resize(num_basis);
  mass.resize(num_basis);
  for (size_t i = 0; i < num_basis; ++i) {
    basis[i] = i;
    auto it = table.find(cpu_atom_symbol[i]);
    if (it == table.end()) {
      PRINT_INPUT_ERROR("Error: no such element << sym << \n");
    }
    mass[i] = it->second;
  }

  label.resize(N);
  for (size_t n = 0; n < N; ++n) {
    size_t atom = n % num_basis;
    label[n] = atom;
  }
}

void Hessian::create_kpoints(const Box& box)
{
  auto dot = [](const std::vector<double>& a, const std::vector<double>& b) -> double {
    return a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
  };
  auto cross = [](const std::vector<double>& a, const std::vector<double>& b) -> std::vector<double> {
    return { a[1]*b[2] - a[2]*b[1],
             a[2]*b[0] - a[0]*b[2],
             a[0]*b[1] - a[1]*b[0] };
  };
  auto transpose = [](const std::vector<std::vector<double>>& m) -> std::vector<std::vector<double>> {
      return { { m[0][0], m[1][0], m[2][0] },
               { m[0][1], m[1][1], m[2][1] },
               { m[0][2], m[1][2], m[2][2] } };
  };
  auto matvec = [](const std::vector<std::vector<double>>& m, const std::vector<double>& v) -> std::vector<double> {
    return { m[0][0]*v[0] + m[0][1]*v[1] + m[0][2]*v[2],
             m[1][0]*v[0] + m[1][1]*v[1] + m[1][2]*v[2],
             m[2][0]*v[0] + m[2][1]*v[1] + m[2][2]*v[2] };
    };
  auto lerp = [](const std::vector<double>& a, const std::vector<double>& b, double t) -> std::vector<double> {
    return { a[0] + t*(b[0] - a[0]),
             a[1] + t*(b[1] - a[1]),
             a[2] + t*(b[2] - a[2]) };
  };
  auto reciprocal_lattice = [&dot,&cross,&transpose](const std::vector<std::vector<double>>& lat) -> std::vector<std::vector<double>> {
    const double volume = dot(lat[0], cross(lat[1], lat[2]));
    std::vector<std::vector<double>> rec(3, std::vector<double>(3));
    rec[0] = cross(lat[1], lat[2]);
    rec[1] = cross(lat[2], lat[0]);
    rec[2] = cross(lat[0], lat[1]);
    for (auto& v : rec)
      for (auto& x : v) x *= 2.0 * M_PI / volume;
    return transpose(rec);
  };

  std::ifstream kin("kpoints.in");
  if (!kin) PRINT_INPUT_ERROR("Cannot open kpoints.in\n");
  std::vector<std::vector<std::vector<double>>> hsps;
  std::vector<std::vector<double>> hsp;
  std::string line;
  while (std::getline(kin, line)) {
    const auto beg = line.find_first_not_of(" \t\r\n");
    if (beg == std::string::npos) {
        if (!hsp.empty()) { hsps.push_back(hsp); hsp.clear(); }
        continue;
    }
    if (line[beg] == '#') continue;
    std::istringstream iss(line);
    double x, y, z;
    if (!(iss >> x >> y >> z)) break;
    hsp.emplace_back(std::vector<double>{x, y, z});
  }
  if (!hsp.empty()) hsps.push_back(hsp);
  num_kpoints = 1 - hsps.size();
  for (const auto& seg : hsps) num_kpoints += seg.size();
  kpath_sym.resize(num_kpoints);

  const std::vector<std::vector<double>> lattice = {
    { box.cpu_h[0] / cx, box.cpu_h[3] / cx, box.cpu_h[6] / cx },
    { box.cpu_h[1] / cy, box.cpu_h[4] / cy, box.cpu_h[7] / cy },
    { box.cpu_h[2] / cz, box.cpu_h[5] / cz, box.cpu_h[8] / cz }
  };
  const auto rec_lat = reciprocal_lattice(lattice);

  std::vector<double> num_interps;
  num_interps.reserve(num_kpoints - 1);
  for (const auto& seg : hsps) {
    for (size_t i = 1; i < seg.size(); ++i) {
      auto start = matvec(rec_lat, seg[i-1]);
      auto end = matvec(rec_lat, seg[i]);
      double dx = end[0] - start[0];
      double dy = end[1] - start[1];
      double dz = end[2] - start[2];
      double dist = std::sqrt(dx*dx + dy*dy + dz*dz);
      num_interps.push_back(static_cast<int>(dist * 100.0));
    }
  }
  for (int n : num_interps) num_kpoints += n;

  kpoints.resize(num_kpoints * 3);
  kpath.resize(num_kpoints);
  std::vector<double> sym_idx;
  size_t k_idx = 0;
  size_t interp_idx = 0;
  double kpath_len = 0.0;

  auto k_first = matvec(rec_lat, hsps[0][0]);
  kpoints[k_idx * 3 + 0] = k_first[0];
  kpoints[k_idx * 3 + 1] = k_first[1];
  kpoints[k_idx * 3 + 2] = k_first[2];
  kpath[k_idx] = kpath_len;
  sym_idx.push_back(k_idx);
  ++k_idx;

  for (const auto& hsp : hsps) {
    for (size_t i = 1; i < hsp.size(); ++i) {
      const auto& start = matvec(rec_lat, hsp[i - 1]);
      const auto& end   = matvec(rec_lat, hsp[i]);
      int n = num_interps[interp_idx++] + 2;

      for (int j = 1; j < n - 1; ++j) {
        double t = static_cast<double>(j) / n;
        auto kpt = lerp(start, end, t);

        kpoints[k_idx * 3 + 0] = kpt[0];
        kpoints[k_idx * 3 + 1] = kpt[1];
        kpoints[k_idx * 3 + 2] = kpt[2];

        double dx = kpt[0] - kpoints[k_idx * 3 - 3];
        double dy = kpt[1] - kpoints[k_idx * 3 - 2];
        double dz = kpt[2] - kpoints[k_idx * 3 - 1];
        kpath_len += std::sqrt(dx * dx + dy * dy + dz * dz);
        kpath[k_idx] = kpath_len;
        ++k_idx;
      }

        // Add the end point
      kpoints[k_idx * 3 + 0] = end[0];
      kpoints[k_idx * 3 + 1] = end[1];
      kpoints[k_idx * 3 + 2] = end[2];

      double dx = end[0] - kpoints[k_idx * 3 - 3];
      double dy = end[1] - kpoints[k_idx * 3 - 2];
      double dz = end[2] - kpoints[k_idx * 3 - 1];
      kpath_len += std::sqrt(dx * dx + dy * dy + dz * dz);
      kpath[k_idx] = kpath_len;
      sym_idx.push_back(k_idx);
      ++k_idx;
    }
  }

  for (size_t kp = 0; kp < kpath_sym.size(); ++kp) {
    kpath_sym[kp] = kpath[sym_idx[kp]];
  }
}

void Hessian::initialize(std::vector<std::string> cpu_atom_symbol, const Box& box, size_t N)
{
  create_basis(cpu_atom_symbol, N);
  create_kpoints(box);
  size_t num_H = num_basis * N * 9;
  size_t num_D = num_basis * num_basis * 9 * num_kpoints;
  H.resize(num_H, 0.0);
  DR.resize(num_D, 0.0);
  if (num_kpoints > 1) // for dispersion calculation
  {
    DI.resize(num_D, 0.0);
  }
}

bool Hessian::is_too_far(
  const Box& box,
  const std::vector<double>& cpu_position_per_atom,
  const size_t n1,
  const size_t n2)
{
  const int number_of_atoms = cpu_position_per_atom.size() / 3;
  double x12 = cpu_position_per_atom[n2] - cpu_position_per_atom[n1];
  double y12 =
    cpu_position_per_atom[n2 + number_of_atoms] - cpu_position_per_atom[n1 + number_of_atoms];
  double z12 = cpu_position_per_atom[n2 + number_of_atoms * 2] -
               cpu_position_per_atom[n1 + number_of_atoms * 2];
  apply_mic(box, x12, y12, z12);
  double d12_square = x12 * x12 + y12 * y12 + z12 * z12;
  return (d12_square > (cutoff * cutoff));
}

void Hessian::find_H(
  Force& force,
  Box& box,
  std::vector<double>& cpu_position_per_atom,
  GPU_Vector<double>& position_per_atom,
  GPU_Vector<int>& type,
  std::vector<Group>& group,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();

  for (size_t nb = 0; nb < num_basis; ++nb) {
    size_t n1 = basis[nb];
    for (size_t n2 = 0; n2 < number_of_atoms; ++n2) {
      if (is_too_far(box, cpu_position_per_atom, n1, n2)) {
        continue;
      }
      size_t offset = (nb * number_of_atoms + n2) * 9;
      find_H12(
        displacement,
        n1,
        n2,
        box,
        position_per_atom,
        type,
        group,
        potential_per_atom,
        force_per_atom,
        virial_per_atom,
        force,
        H.data() + offset);
    }
  }
}

static void find_exp_ikr(
  const size_t n1,
  const size_t n2,
  const double* k,
  const Box& box,
  const std::vector<double>& cpu_position_per_atom,
  double& cos_kr,
  double& sin_kr)
{
  const int number_of_atoms = cpu_position_per_atom.size() / 3;
  double x12 = cpu_position_per_atom[n2] - cpu_position_per_atom[n1];
  double y12 =
    cpu_position_per_atom[n2 + number_of_atoms] - cpu_position_per_atom[n1 + number_of_atoms];
  double z12 = cpu_position_per_atom[n2 + number_of_atoms * 2] -
               cpu_position_per_atom[n1 + number_of_atoms * 2];
  apply_mic(box, x12, y12, z12);
  double kr = k[0] * x12 + k[1] * y12 + k[2] * z12;
  cos_kr = cos(kr);
  sin_kr = sin(kr);
}

void Hessian::output_D()
{
  FILE* fid = fopen("D.out", "w");
  for (size_t nk = 0; nk < num_kpoints; ++nk) {
    size_t offset = nk * num_basis * num_basis * 9;
    for (size_t n1 = 0; n1 < num_basis * 3; ++n1) {
      for (size_t n2 = 0; n2 < num_basis * 3; ++n2) {
        // cuSOLVER requires column-major
        fprintf(fid, "%g ", DR[offset + n1 + n2 * num_basis * 3]);
      }
      if (num_kpoints > 1) {
        for (size_t n2 = 0; n2 < num_basis * 3; ++n2) {
          // cuSOLVER requires column-major
          fprintf(fid, "%g ", DI[offset + n1 + n2 * num_basis * 3]);
        }
      }
      fprintf(fid, "\n");
    }
  }
  fclose(fid);
}

void Hessian::find_omega(FILE* fid, size_t offset, size_t nk)
{
  size_t dim = num_basis * 3;
  std::vector<double> W(dim);
  eig_hermitian_QR(dim, DR.data() + offset, DI.data() + offset, W.data());
  double natural_to_THz = 1.0e6 / (TIME_UNIT_CONVERSION * TIME_UNIT_CONVERSION);
  fprintf(fid, "%.6f ", kpath[nk]);
  for (size_t n = 0; n < dim; ++n) {
    fprintf(fid, "%g ", W[n] * natural_to_THz);
  }
  fprintf(fid, "\n");
}

void Hessian::find_omega_batch(FILE* fid)
{
  size_t dim = num_basis * 3;
  std::vector<double> W(dim * num_kpoints);
  eig_hermitian_Jacobi_batch(dim, num_kpoints, DR.data(), DI.data(), W.data());
  double natural_to_THz = 1.0e6 / (TIME_UNIT_CONVERSION * TIME_UNIT_CONVERSION);
  for (size_t nk = 0; nk < num_kpoints; ++nk) {
    size_t offset = nk * dim;
    fprintf(fid, "%.6f ", kpath[nk]);
    for (size_t n = 0; n < dim; ++n) {
      fprintf(fid, "%g ", W[offset + n] * natural_to_THz);
    }
    fprintf(fid, "\n");
  }
}

void Hessian::find_dispersion(const Box& box, const std::vector<double>& cpu_position_per_atom)
{
  const int number_of_atoms = cpu_position_per_atom.size() / 3;

  FILE* fid_omega2 = fopen("omega2.out", "w");
  fprintf(fid_omega2, "# ");
  for (size_t i = 0; i < kpath_sym.size(); ++i) {
    fprintf(fid_omega2, " %.6f", kpath_sym[i]);
    }
  fprintf(fid_omega2, "\n");
  for (size_t nk = 0; nk < num_kpoints; ++nk) {
    size_t offset = nk * num_basis * num_basis * 9;
    for (size_t nb = 0; nb < num_basis; ++nb) {
      size_t n1 = basis[nb];
      size_t label_1 = label[n1];
      double mass_1 = mass[label_1];
      for (size_t n2 = 0; n2 < number_of_atoms; ++n2) {
        if (is_too_far(box, cpu_position_per_atom, n1, n2))
          continue;
        double cos_kr, sin_kr;
        find_exp_ikr(n1, n2, kpoints.data() + nk * 3, box, cpu_position_per_atom, cos_kr, sin_kr);

        size_t label_2 = label[n2];
        double mass_2 = mass[label_2];
        double mass_factor = 1.0 / sqrt(mass_1 * mass_2);
        double* H12 = H.data() + (nb * number_of_atoms + n2) * 9;
        for (size_t a = 0; a < 3; ++a) {
          for (size_t b = 0; b < 3; ++b) {
            size_t a3b = a * 3 + b;
            size_t row = label_1 * 3 + a;
            size_t col = label_2 * 3 + b;
            // cuSOLVER requires column-major
            size_t index = offset + col * num_basis * 3 + row;
            DR[index] += H12[a3b] * cos_kr * mass_factor;
            DI[index] += H12[a3b] * sin_kr * mass_factor;
          }
        }
      }
    }
    if (num_basis > 10) {
      find_omega(fid_omega2, offset, nk);
    } // > 32x32
  }
  output_D();
  if (num_basis <= 10) {
    find_omega_batch(fid_omega2);
  } // <= 32x32
  fclose(fid_omega2);
}

void Hessian::find_D(const Box& box, std::vector<double>& cpu_position_per_atom)
{
  const int number_of_atoms = cpu_position_per_atom.size() / 3;

  for (size_t nb = 0; nb < num_basis; ++nb) {
    size_t n1 = basis[nb];
    size_t label_1 = label[n1];
    double mass_1 = mass[label_1];
    for (size_t n2 = 0; n2 < number_of_atoms; ++n2) {
      if (is_too_far(box, cpu_position_per_atom, n1, n2)) {
        continue;
      }

      size_t label_2 = label[n2];
      double mass_2 = mass[label_2];
      double mass_factor = 1.0 / sqrt(mass_1 * mass_2);
      double* H12 = H.data() + (nb * number_of_atoms + n2) * 9;
      for (size_t a = 0; a < 3; ++a) {
        for (size_t b = 0; b < 3; ++b) {
          size_t a3b = a * 3 + b;
          size_t row = label_1 * 3 + a;
          size_t col = label_2 * 3 + b;
          // cuSOLVER requires column-major
          size_t index = col * num_basis * 3 + row;
          DR[index] += H12[a3b] * mass_factor;
        }
      }
    }
  }
}

void Hessian::find_eigenvectors()
{
  std::ofstream eigfile;
  eigfile.open("eigenvector.out", std::ios::out | std::ios::binary);

  size_t dim = num_basis * 3;
  std::vector<double> W(dim);
  std::vector<double> eigenvectors(dim * dim);
  eigenvectors_symmetric_Jacobi(dim, DR.data(), W.data(), eigenvectors.data());

  double natural_to_THz = 1.0e6 / (TIME_UNIT_CONVERSION * TIME_UNIT_CONVERSION);

  // output eigenvalues
  float om2;
  for (size_t n = 0; n < dim; n++) {
    om2 = (float)(W[n] * natural_to_THz);
    eigfile.write((char*)&om2, sizeof(float));
  }

  // output eigenvectors
  float eig;
  for (size_t col = 0; col < dim; col++) {
    for (size_t a = 0; a < 3; a++) {
      for (size_t b = 0; b < num_basis; b++) {
        size_t row = a + b * 3;
        // column-major order from cuSolver
        eig = (float)eigenvectors[row + col * dim];
        eigfile.write((char*)&eig, sizeof(float));
      }
    }
  }
  eigfile.close();
}

void Hessian::parse(const char** param, size_t num_param)
{
  if (num_param != 3) {
    PRINT_INPUT_ERROR("compute_phonon should have 2 parameters.\n");
  }
  // cutoff
  if (!is_valid_real(param[1], &cutoff)) {
    PRINT_INPUT_ERROR("cutoff for compute_phonon should be a number.\n");
  }
  if (cutoff <= 0) {
    PRINT_INPUT_ERROR("cutoff for compute_phonon should be positive.\n");
  }
  printf("Cutoff distance for compute_phonon = %g A.\n", cutoff);

  // displacement
  if (!is_valid_real(param[2], &displacement)) {
    PRINT_INPUT_ERROR("displacement for compute_phonon should be a number.\n");
  }
  if (displacement <= 0) {
    PRINT_INPUT_ERROR("displacement for compute_phonon should be positive.\n");
  }
  printf("displacement for compute_phonon = %g A.\n", displacement);
}
